#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <string>
#include <cmath>
#include <cassert>
#include <map>
#include <unistd.h>

#include "pathfinder_common.h"
#include "scene.h"
#include "solver.h"

using namespace std;

void gpuCheckError( hipError_t err,
                         const char *file,
                         int line ) {
   if (err != hipSuccess) {
       printf( "%s in %s at line %d\n", hipGetErrorString( err ),
              file, line );
       exit( EXIT_FAILURE );
   }
}

static int get_max_threads()
{
    hipDeviceProp_t prop;
    int count, maxThreads = 0;
    GPU_CHECKERROR(hipGetDeviceCount(&count));
    int device = 0;
    for (int i = 0; i < count; i++) {
        GPU_CHECKERROR(hipGetDeviceProperties(&prop, i));
        if (prop.maxThreadsPerBlock > maxThreads) {
            maxThreads = prop.maxThreadsPerBlock;
            device = i;
        }
    }
    assert(maxThreads != 0);
    GPU_CHECKERROR(hipSetDevice(device));
    return maxThreads;
}

static scene_t scene;
// Make a map to store human references
static map<string, human_t *> humanMap;
// Make a map to store goal references
static map<string, point_t *> goalMap;
// We need to serialize the scene
static point_t *linearGrid;

int main(int argc, char** argv)
{
	if (argc != 3) {
		fprintf(stderr, "usage: %s <scene.map> <human.dsc>", *argv);
		exit(1);
	}
	FILE *mapFile = fopen(argv[1], "r");
	if (!mapFile) {
		perror(argv[1]);
		return ENOFILE;
	}
	buildMap(mapFile, &scene, &humanMap, &goalMap);
	// We're done with the file, so close it
	fclose(mapFile);

	// Make the grid linear
	linearizeGrid(&scene, &linearGrid);
	// Stick the humans in an array
	int numHumans = humanMap.size();
	human_t *humans = (human_t *) malloc(numHumans * sizeof(human_t));
	int pos = 0;
	for (map<string, human_t *>::iterator it = humanMap.begin(); it != humanMap.end(); ++it)
		humans[pos++] = *(it->second);

	// Copy over the grid, humans, and statistics
	point_t *d_linearGrid;
	human_t *d_humans;
	stat_t *d_stats;
	// Allocate space to store the results
	void *results, *d_results;
	int *d_remainingHumans;
	unsigned int *d_itrCnt, itrCnt, zero = 0;
	GPU_CHECKERROR(hipMalloc(&d_linearGrid, sizeof(point_t) * scene.width * scene.height));
	GPU_CHECKERROR(hipMalloc(&d_humans, sizeof(human_t) * numHumans));
	GPU_CHECKERROR(hipMalloc(&d_stats, sizeof(stat_t) * numHumans));
	GPU_CHECKERROR(hipMalloc(&d_remainingHumans, sizeof(int)));
	GPU_CHECKERROR(hipMalloc(&d_itrCnt, sizeof(int)));
	GPU_CHECKERROR(hipHostAlloc(&results, scene.width * scene.height * sizeof(simple_point_t) * numHumans + sizeof(int) * numHumans * 2, hipHostMallocMapped));
	GPU_CHECKERROR(hipHostGetDevicePointer(&d_results, results, 0));
	debugPrintf("pinned memory: %lu, width: %d, height: %d\n", scene.width * scene.height * sizeof(simple_point_t) * numHumans + sizeof(int) * numHumans * 2, scene.width, scene.height);
	GPU_CHECKERROR(hipMemcpy(d_linearGrid, linearGrid, sizeof(point_t) * scene.width * scene.height, hipMemcpyHostToDevice));
	GPU_CHECKERROR(hipMemcpy(d_humans, humans, sizeof(human_t) * numHumans, hipMemcpyHostToDevice));
	GPU_CHECKERROR(hipMemcpy(d_remainingHumans, &numHumans, sizeof(int), hipMemcpyHostToDevice));
	GPU_CHECKERROR(hipMemcpy(d_itrCnt, &zero, sizeof(int), hipMemcpyHostToDevice));

	// Get the maximum number of threads
	int threads = get_max_threads();
	int blocks = (int) ceil(((double) numHumans) / threads);
	// Increase the heap size
	GPU_CHECKERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, 200 * 1024 * 1024));
	// Launch the kernel
	solveScene<<<blocks, threads>>>(d_linearGrid, d_humans, d_stats, scene.width, scene.height, numHumans, d_remainingHumans, d_results, d_itrCnt);
	GPU_CHECKERROR(hipMemcpy(&itrCnt, d_itrCnt, sizeof(int), hipMemcpyDeviceToHost));
	GPU_CHECKERROR(hipDeviceSynchronize());

	// Free the memory
	GPU_CHECKERROR(hipFree(d_linearGrid));
	GPU_CHECKERROR(hipFree(d_humans));
	GPU_CHECKERROR(hipFree(d_stats));
	GPU_CHECKERROR(hipFree(d_remainingHumans));

	// Now, analyze the results
	analyzeResults(results, &humanMap, &scene, itrCnt, humans);
	GPU_CHECKERROR(hipHostFree(results));
	free(linearGrid);
	free(humans);
	freeScene(&scene, &humanMap);

	return 0;
}
